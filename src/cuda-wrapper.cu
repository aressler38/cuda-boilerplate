#include "hip/hip_runtime.h"
#include "cuda-wrapper.h"

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void callAComplexTypeKernel(float *data, unsigned size) {
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size) {
		complex_t z(1.0, index);
		data[index] = z.magnitude();
	}
}


/**
 * Host function that copies the data and launches the work on GPU
 */
float *callAComplexType(float *data, unsigned size) {
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	float *rc = new float[size];
	float *gpuData;

	hipMalloc((void **)&gpuData, sizeof(float)*size);
	hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice);

	callAComplexTypeKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost);
	hipFree(gpuData);
	return rc;
}
